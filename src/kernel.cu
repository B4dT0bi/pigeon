#include "hip/hip_runtime.h"

//__HIPCC__ defines whether nvcc is steering compilation or not
//__CUDA_ARCH__is always undefined when compiling host code, steered by nvcc or not
//__CUDA_ARCH__is only defined for the device code trajectory of compilation steered by nvcc

#include "hip/hip_runtime.h"
#include ""



#include "platform.h"
#include "defs.h"
#include "bits.h"
#include "position.h"
#include "movelist.h"
#include "eval.h"
#include "table.h"

#include <stdio.h>



__device__ void Foo()
{
    int i = threadIdx.x;

    Pigeon::Position pos;
    pos.Reset();

    Pigeon::MoveList moves;
    moves.FindMoves( pos );

    printf( "Thread %d says %d\n", i, moves.mCount );
}

__global__ void RunFoo()
{
    Foo();
}

using namespace Pigeon;

#include "negamax.h"

int main()
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    //cudaStatus = hipSetDevice(0);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    //    goto Error;
    //}

    RunFoo<<< 1, 1 >>>();


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        //goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        //goto Error;
    }



    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    // Allocate GPU buffers for three vectors (two input, one output)    .
    //cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}
    //
    //cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}
    //
    //cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}
    //
    //// Copy input vectors from host memory to GPU buffers.
    //cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}
    //
    //cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}
    // Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    return 0;
}
