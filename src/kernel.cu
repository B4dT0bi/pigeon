#include "hip/hip_runtime.h"
// kernel.cu - PIGEON CHESS ENGINE (c) 2012-2016 Stuart Riffle

#include "platform.h"
#include "defs.h"
#include "bits.h"
#include "simd.h"
#include "position.h"
#include "movelist.h"
#include "eval.h"
#include "table.h"
#include "search.h"


__global__ void SearchPositionsOnGPU( const Pigeon::SearchJobInput* inputBuf, Pigeon::SearchJobOutput* outputBuf, int count, Pigeon::HashTable* hashTable, Pigeon::Evaluator* evaluator )
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if( idx >= count )
		return;
		 
	const Pigeon::SearchJobInput*	input	= inputBuf  + idx;
	Pigeon::SearchJobOutput*		output	= outputBuf + idx;
	Pigeon::SearchMetrics			metrics;

	Pigeon::SearchState< 1, Pigeon::u64 > ss;

    ss.mHashTable	= hashTable;
    ss.mEvaluator	= evaluator;
    ss.mMetrics		= &metrics;

    output->mScore = ss.RunToDepth( input->mPosition, input->mSearchDepth );
	output->mNodes = metrics.mNodesTotal;

    ss.ExtractBestLine( &output->mBestLine );

	__threadfence();
}


void QueueSearchBatch( Pigeon::SearchBatch* batch, int blockSize )
{
	// Copy the inputs to device

	hipMemcpyAsync( batch->mInputDev, batch->mInputHost, sizeof( Pigeon::SearchJobInput ) * batch->mCount, hipMemcpyHostToDevice, batch->mStream );

	// Clear the device outputs

	hipMemsetAsync( batch->mOutputDev, 0, sizeof( Pigeon::SearchJobOutput ) * batch->mCount, batch->mStream );

	// Run the search kernel

	int blockCount = (batch->mCount + blockSize - 1) / blockSize;
	SearchPositionsOnGPU<<< blockCount, blockSize, 0, batch->mStream >>>( batch->mInputDev, batch->mOutputDev, batch->mCount, batch->mHashTable, batch->mEvaluator );

	// Copy the outputs to host

	hipMemcpyAsync( batch->mOutputHost, batch->mOutputDev, sizeof( Pigeon::SearchJobOutput ) * batch->mCount, hipMemcpyDeviceToHost, batch->mStream );

	// Record an event we can test for completion

	hipEventRecord( batch->mEvent, batch->mStream );
}

